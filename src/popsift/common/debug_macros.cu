/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "debug_macros.h"

#include <assert.h>

using namespace std;

void pop_check_last_error( const char* file, size_t line )
{
    hipError_t err = hipGetLastError( );
    if( err != hipSuccess ) {
        std::cerr << __FILE__ << ":" << __LINE__ << std::endl
                  << "    called from " << file << ":" << line << std::endl
                  << "    hipGetLastError failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
}

namespace popsift { namespace cuda {
void malloc_dev( void** ptr, int sz,
                 const char* file, int line )
{
    hipError_t err;
    err = hipMalloc( ptr, sz );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
#ifdef DEBUG_INIT_DEVICE_ALLOCATIONS
    popsift::cuda::memset_sync( *ptr, 0, sz, file, line );
#endif // NDEBUG
}
} }

namespace popsift { namespace cuda {
void malloc_hst( void** ptr, int sz,
                 const char* file, int line )
{
    hipError_t err;
    err = hipHostMalloc( ptr, sz );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipHostMalloc failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
#ifdef DEBUG_INIT_DEVICE_ALLOCATIONS
    memset( *ptr, 0, sz );
#endif // NDEBUG
}
} }

namespace popsift { namespace cuda {
void memcpy_async( void* dst, const void* src, size_t sz,
                   hipMemcpyKind type, hipStream_t stream,
                   const char* file, size_t line )
{
    POP_CHECK_NON_NULL_FL( dst, "Dest ptr in memcpy async is null.", file, line );
    POP_CHECK_NON_NULL_FL( src, "Source ptr in memcpy async is null.", file, line );
    POP_CHECK_NON_NULL_FL( sz, "Size in memcpy async is null.", file, line );

    hipError_t err;
    err = hipMemcpyAsync( dst, src, sz, type, stream );
    if( err != hipSuccess ) {
        cerr << file << ":" << line << endl
             << "    " << "Failed to copy "
             << (type==hipMemcpyHostToDevice?"host-to-device":"device-to-host")
             << ": ";
        cerr << hipGetErrorString(err) << endl;
        cerr << "    src ptr=" << hex << (size_t)src << dec << endl
             << "    dst ptr=" << hex << (size_t)dst << dec << endl;
        exit( -__LINE__ );
    }
    POP_CUDA_FATAL_TEST( err, "Failed to copy host-to-device: " );
}

void memcpy_sync( void* dst, const void* src, size_t sz, hipMemcpyKind type, const char* file, size_t line )
{
    POP_CHECK_NON_NULL( dst, "Dest ptr in memcpy async is null." );
    POP_CHECK_NON_NULL( src, "Source ptr in memcpy async is null." );
    POP_CHECK_NON_NULL( sz, "Size in memcpy async is null." );

    hipError_t err;
    err = hipMemcpy( dst, src, sz, type );
    if( err != hipSuccess ) {
        cerr << "    " << "Failed to copy "
             << (type==hipMemcpyHostToDevice?"host-to-device":"device-to-host")
             << ": ";
        cerr << hipGetErrorString(err) << endl;
        cerr << "    src ptr=" << hex << (size_t)src << dec << endl
             << "    dst ptr=" << hex << (size_t)dst << dec << endl;
        exit( -__LINE__ );
    }
    POP_CUDA_FATAL_TEST( err, "Failed to copy host-to-device: " );
}

void memset_async( void* ptr, int value, size_t bytes, hipStream_t stream, const char* file, size_t line )
{
    hipError_t err;
    err = hipMemsetAsync( ptr, value, bytes, stream );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipMemsetAsync failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
}

void memset_sync( void* ptr, int value, size_t bytes, const char* file, size_t line )
{
    hipError_t err;
    err = hipMemset( ptr, value, bytes );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipMemset failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
}
} }

namespace popsift { namespace cuda {
hipStream_t stream_create( const char* file, size_t line )
{
    hipStream_t stream;
    hipError_t err;
    err = hipStreamCreate( &stream );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipStreamCreate failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
    return stream;
}
void stream_destroy( hipStream_t s, const char* file, size_t line )
{
    hipError_t err;
    err = hipStreamDestroy( s );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipStreamDestroy failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
}
hipEvent_t event_create( const char* file, size_t line )
{
    hipEvent_t ev;
    hipError_t err;
    err = hipEventCreate( &ev );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipEventCreate failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
    return ev;
}
void event_destroy( hipEvent_t ev, const char* file, size_t line )
{
    hipError_t err;
    err = hipEventDestroy( ev );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipEventDestroy failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
}
void event_record( hipEvent_t ev, hipStream_t s, const char* file, size_t line )
{
    hipError_t err;
    err = hipEventRecord( ev, s );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipEventRecord failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
}
void event_wait( hipEvent_t ev, hipStream_t s, const char* file, size_t line )
{
    hipError_t err;
    err = hipStreamWaitEvent( s, ev, 0 );
    if( err != hipSuccess ) {
        std::cerr << file << ":" << line << std::endl
                  << "    hipStreamWaitEvent failed: " << hipGetErrorString(err) << std::endl;
        exit( -__LINE__ );
    }
}

float event_diff( hipEvent_t from, hipEvent_t to )
{   
    float ms;
    hipEventElapsedTime( &ms, from, to );
    return ms;
}

} // namespace cuda
} // namespace popsift

