/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "debug_macros.h"

#include <cassert>

using namespace std;

void pop_sync_check_last_error( const char* file, size_t line )
{
    hipDeviceSynchronize();
    pop_check_last_error( file, line );
}

void pop_check_last_error( const char* file, size_t line )
{
    hipError_t err = hipGetLastError( );
    POP_CUDA_FATAL_TEST(err, "hipGetLastError failed: ");
}

namespace popsift { namespace cuda {
void malloc_dev( void** ptr, int sz,
                 const char* file, int line )
{
    hipError_t err;
    err = hipMalloc( ptr, sz );
    POP_CUDA_FATAL_TEST(err, "hipMalloc failed: ");
#ifdef DEBUG_INIT_DEVICE_ALLOCATIONS
    popsift::cuda::memset_sync( *ptr, 0, sz, file, line );
#endif // NDEBUG
}
} }

namespace popsift { namespace cuda {
void malloc_hst( void** ptr, int sz,
                 const char* file, int line )
{
    hipError_t err;
    err = hipHostMalloc( ptr, sz );
    POP_CUDA_FATAL_TEST(err, "hipHostMalloc failed: ");
#ifdef DEBUG_INIT_DEVICE_ALLOCATIONS
    memset( *ptr, 0, sz );
#endif // NDEBUG
}
} }

namespace popsift { namespace cuda {
void memcpy_async( void* dst, const void* src, size_t sz,
                   hipMemcpyKind type, hipStream_t stream,
                   const char* file, size_t line )
{
    POP_CHECK_NON_NULL_FL( dst, "Dest ptr in memcpy async is null.", file, line );
    POP_CHECK_NON_NULL_FL( src, "Source ptr in memcpy async is null.", file, line );
    POP_CHECK_NON_NULL_FL( sz, "Size in memcpy async is null.", file, line );

    hipError_t err;
    err = hipMemcpyAsync( dst, src, sz, type, stream );
    if( err != hipSuccess ) {
        std::stringstream ss;
        ss << "Failed to copy " << (type == hipMemcpyHostToDevice ? "host-to-device" : "device-to-host") << ": ";
        ss << hipGetErrorString(err) << endl;
        ss << "    src ptr=" << hex << (size_t)src << dec << endl
           << "    dst ptr=" << hex << (size_t)dst << dec << endl;
        POP_FATAL(ss.str());
    }
}

void memcpy_sync( void* dst, const void* src, size_t sz, hipMemcpyKind type, const char* file, size_t line )
{
    POP_CHECK_NON_NULL( dst, "Dest ptr in memcpy async is null." );
    POP_CHECK_NON_NULL( src, "Source ptr in memcpy async is null." );
    POP_CHECK_NON_NULL( sz, "Size in memcpy async is null." );

    hipError_t err;
    err = hipMemcpy( dst, src, sz, type );
    if( err != hipSuccess ) {
        std::stringstream ss;
        ss << "Failed to copy " << (type == hipMemcpyHostToDevice ? "host-to-device" : "device-to-host") << ": ";
        ss << hipGetErrorString(err) << endl;
        ss << "    src ptr=" << hex << (size_t)src << dec << endl
           << "    dst ptr=" << hex << (size_t)dst << dec << endl;
        POP_FATAL(ss.str())
    }
}

void memset_async( void* ptr, int value, size_t bytes, hipStream_t stream, const char* file, size_t line )
{
    hipError_t err;
    err = hipMemsetAsync( ptr, value, bytes, stream );
    POP_CUDA_FATAL_TEST(err, "hipMemsetAsync failed: ");
}

void memset_sync( void* ptr, int value, size_t bytes, const char* file, size_t line )
{
    hipError_t err;
    err = hipMemset( ptr, value, bytes );
    POP_CUDA_FATAL_TEST(err, "hipMemset failed: ");
}
} }

namespace popsift { namespace cuda {
hipStream_t stream_create( const char* file, size_t line )
{
    hipStream_t stream;
    hipError_t err;
    err = hipStreamCreate( &stream );
    POP_CUDA_FATAL_TEST(err, "hipStreamCreate failed: ");
    return stream;
}
void stream_destroy( hipStream_t s, const char* file, size_t line )
{
    hipError_t err;
    err = hipStreamDestroy( s );
    POP_CUDA_FATAL_TEST(err, "hipStreamDestroy failed: ");
}
hipEvent_t event_create( const char* file, size_t line )
{
    hipEvent_t ev;
    hipError_t err;
    err = hipEventCreate( &ev );
    POP_CUDA_FATAL_TEST(err, "hipEventCreate failed: ");
    return ev;
}
void event_destroy( hipEvent_t ev, const char* file, size_t line )
{
    hipError_t err;
    err = hipEventDestroy( ev );
    POP_CUDA_FATAL_TEST(err, "hipEventDestroy failed: ");
}
void event_record( hipEvent_t ev, hipStream_t s, const char* file, size_t line )
{
    hipError_t err;
    err = hipEventRecord( ev, s );
    POP_CUDA_FATAL_TEST(err, "hipEventRecord failed: ");
}
void event_wait( hipEvent_t ev, hipStream_t s, const char* file, size_t line )
{
    hipError_t err;
    err = hipStreamWaitEvent( s, ev, 0 );
    POP_CUDA_FATAL_TEST(err, "hipStreamWaitEvent failed: ");
}

float event_diff( hipEvent_t from, hipEvent_t to )
{
    float ms;
    hipEventElapsedTime( &ms, from, to );
    return ms;
}

} // namespace cuda
} // namespace popsift

