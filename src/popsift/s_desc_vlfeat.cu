#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *           2018-2020, University of Oslo
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "popsift/sift_config.h"

#include "common/assist.h"
#include "common/debug_macros.h"
#include "common/vec_macros.h"
#include "s_desc_vlfeat.h"
#include "s_gradiant.h"
#include "sift_constants.h"
#include "sift_pyramid.h"

#include <cstdio>

using namespace popsift;

__device__ static inline
void ext_desc_vlfeat_sub( const float         ang,
                          const Extremum*     ext,
                          float* __restrict__ features,
                          hipTextureObject_t layer_tex,
                          const int           width,
                          const int           height )
{
    const float x     = ext->xpos;
    const float y     = ext->ypos;
    const int   level = ext->lpos; // old_level;
    const float sig   = ext->sigma;
    const float SBP   = fabsf(DESC_MAGNIFY * sig);

    if( SBP == 0 ) {
        return;
    }

    float cos_t;
    float sin_t;
    __sincosf( ang, &sin_t, &cos_t );

    const float csbp  = cos_t * SBP;
    const float ssbp  = sin_t * SBP;
    const float crsbp = cos_t / SBP;
    const float srsbp = sin_t / SBP;

    // We have 4x4*16 bins.
    // There centers have the offsets -1.5, -0.5, 0.5, 1.5 from the
    // keypoint. The points that support them stretch from -2 to 2
    const float2 maxdist = make_float2( -2.0f, -2.0f );

    // We rotate the corner of the maximum range by the keypoint orientation.
    // const float ptx = csbp * maxdist - ssbp * maxdist;
    // const float pty = csbp * maxdist + ssbp * maxdist;
    const float ptx = fabsf( ::fmaf( csbp, maxdist.x, -ssbp * maxdist.y ) );
    const float pty = fabsf( ::fmaf( csbp, maxdist.y,  ssbp * maxdist.x ) );

    const float bsz = 2.0f * ( fabsf(csbp) + fabsf(ssbp) );

    const int   xmin = max(1,          (int)floorf(x - ptx - bsz));
    const int   ymin = max(1,          (int)floorf(y - pty - bsz));
    const int   xmax = min(width - 2,  (int)floorf(x + ptx + bsz));
    const int   ymax = min(height - 2, (int)floorf(y + pty + bsz));

    __shared__ float dpt[128];

#if POPSIFT_IS_DEFINED(POPSIFT_HAVE_COOPERATIVE_GROUPS)
    cg::thread_block          block = cg::this_thread_block();
    cg::thread_block_tile<32> tile  = cg::tiled_partition<32>( block );

    for( int i=tile.thread_rank(); i<128; i+=tile.size() )
    {
        dpt[i] = 0.0f;
    }

    tile.sync();
#else
    for( int i=threadIdx.x; i<128; i+=blockDim.x )
    {
        dpt[i] = 0.0f;
    }

    __syncthreads();
#endif

    for( int pix_y = ymin; pix_y <= ymax; pix_y += 1 )
    {
        for( int base_x = xmin; base_x <= xmax; base_x += 32 )
        {
            float mod;
            float th;

#if POPSIFT_IS_DEFINED(POPSIFT_HAVE_COOPERATIVE_GROUPS)
            get_gradiant32( tile, mod, th, base_x, pix_y, layer_tex, level );
#else
            get_gradiant32( mod, th, base_x, pix_y, layer_tex, level );
#endif

            mod /= 2.0f; // Our mod is double that of vlfeat. Huh.

            th -= ang;
            while( th > M_PI2 ) th -= M_PI2;
            while( th < 0.0f  ) th += M_PI2;
#if POPSIFT_IS_DEFINED(POPSIFT_HAVE_COOPERATIVE_GROUPS)
            tile.sync();

            const int pix_x = base_x + tile.thread_rank();
#else
            __syncthreads();

            const int pix_x = base_x + threadIdx.x;
#endif

            if( ( pix_y <= ymax ) && ( pix_x <= xmax ) )
            {
#if POPSIFT_IS_DEFINED(POPSIFT_HAVE_COOPERATIVE_GROUPS)
                cg::coalesced_group co_tile = cg::coalesced_threads();
#endif

                // d : distance from keypoint
                const float2 d = make_float2( pix_x - x, pix_y - y );

                // n : normalized distance from keypoint
                const float2 n = make_float2( ::fmaf( crsbp, d.x,  srsbp * d.y ),
                                              ::fmaf( crsbp, d.y, -srsbp * d.x ) ); 

                const float  ww = __expf( -scalbnf(n.x*n.x + n.y*n.y, -3));

                const float nt = 8.0f * th / M_PI2;

                // neighbouring tile on the lower side: -2, -1, 0 or 1
                // (must use floorf because casting rounds towards zero
                const int3 t0 = make_int3( (int)floorf(n.x - 0.5f),
                                           (int)floorf(n.y - 0.5f),
                                           (int)nt );
                const float wgt_x = - ( n.x - ( t0.x + 0.5f ) );
                const float wgt_y = - ( n.y - ( t0.y + 0.5f ) );
                const float wgt_t = - ( nt  - t0.z );

                for( int tx=0; tx<2; tx++ )
                {
                    for( int ty=0; ty<2; ty++ )
                    {
                        for( int tt=0; tt<2; tt++ )
                        {
                            if( ( t0.y + ty >= -2 ) &&
                                ( t0.y + ty <   2 ) &&
                                ( t0.x + tx >= -2 ) &&
                                ( t0.x + tx <   2 ) )
                            {
                                float i_wgt_x = ( tx == 0 ) ? 1.0f + wgt_x : wgt_x;
                                float i_wgt_y = ( ty == 0 ) ? 1.0f + wgt_y : wgt_y;
                                float i_wgt_t = ( tt == 0 ) ? 1.0f + wgt_t : wgt_t;

                                i_wgt_x = fabsf( i_wgt_x );
                                i_wgt_y = fabsf( i_wgt_y );
                                i_wgt_t = fabsf( i_wgt_t );

                                const float val = ww
                                                * mod
                                                * i_wgt_x
                                                * i_wgt_y
                                                * i_wgt_t;

                                const int offset =  80
                                                + ( t0.y + ty ) * 32
                                                + ( t0.x + tx ) * 8
                                                + ( t0.z + tt ) % 8;

                                atomicAdd( &dpt[offset], val );
                            }

#if POPSIFT_IS_DEFINED(POPSIFT_HAVE_COOPERATIVE_GROUPS)
                            co_tile.sync();
#else
                            // cannot be done before CUDA 9
#endif
                        }
                    }
                }
            }
#if POPSIFT_IS_DEFINED(POPSIFT_HAVE_COOPERATIVE_GROUPS)
            tile.sync();
#else
            __syncthreads();
#endif
        }
    }

#if POPSIFT_IS_DEFINED(POPSIFT_HAVE_COOPERATIVE_GROUPS)
    for( int i=tile.thread_rank(); i<128; i+=tile.size() )
    {
        features[i] = dpt[i];
    }
#else
    for( int i=threadIdx.x; i<128; i+=blockDim.x )
    {
        features[i] = dpt[i];
    }
#endif
}

__global__ void ext_desc_vlfeat(int octave, hipTextureObject_t layer_tex, int w, int h)
{           
    const int   o_offset =  dct.ori_ps[octave] + blockIdx.x;
    Descriptor* desc     = &dbuf.desc           [o_offset];
    const int   ext_idx  =  dobuf.feat_to_ext_map[o_offset];
    Extremum*   ext      =  dobuf.extrema + ext_idx;

    const int   ext_base =  ext->idx_ori;
    const int   ori_num  =  o_offset - ext_base;
    const float ang      =  ext->orientation[ori_num];

    ext_desc_vlfeat_sub( ang,
                         ext,
                         desc->features,
                         layer_tex,
                         w,
                         h );
}

namespace popsift
{

bool start_ext_desc_vlfeat( const int octave, Octave& oct_obj )
{
    dim3 block;
    dim3 grid;
    grid.x = hct.ori_ct[octave];
    grid.y = 1;
    grid.z = 1;

    if( grid.x == 0 ) return false;

    block.x = 32;
    block.y = 1;
    block.z = 1;

    size_t shared_size = 4 * 128 * sizeof(float);

    ext_desc_vlfeat
        <<<grid,block,shared_size,oct_obj.getStream()>>>
        ( octave,
          oct_obj.getDataTexPoint( ),
          oct_obj.getWidth(),
          oct_obj.getHeight() );

    POP_SYNC_CHK;

    return true;
}

}; // namespace popsift

