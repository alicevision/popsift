#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *           2018-2020, University of Oslo
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "common/assist.h"
#include "common/debug_macros.h"
#include "common/vec_macros.h"
#include "s_desc_vlfeat.h"
#include "s_gradiant.h"
#include "sift_constants.h"
#include "sift_pyramid.h"

#include <cstdio>

using namespace popsift;

__device__ static inline
void ext_desc_vlfeat_sub( const float         ang,
                          const Extremum*     ext,
                          float* __restrict__ features,
                          hipTextureObject_t layer_tex,
                          const int           width,
                          const int           height )
{
    const float x     = ext->xpos;
    const float y     = ext->ypos;
    const int   level = ext->lpos; // old_level;
    const float sig   = ext->sigma;
    const float SBP   = fabsf(DESC_MAGNIFY * sig);

    if( SBP == 0 ) {
        return;
    }

    float cos_t;
    float sin_t;
    __sincosf( ang, &sin_t, &cos_t );

    const float csbp  = cos_t * SBP;
    const float ssbp  = sin_t * SBP;
    const float crsbp = cos_t / SBP;
    const float srsbp = sin_t / SBP;

    // We have 4x4*16 bins.
    // There centers have the offsets -1.5, -0.5, 0.5, 1.5 from the
    // keypoint. The points that support them stretch from -2 to 2
    const float2 maxdist = make_float2( -2.0f, -2.0f );

    // We rotate the corner of the maximum range by the keypoint orientation.
    // const float ptx = csbp * maxdist - ssbp * maxdist;
    // const float pty = csbp * maxdist + ssbp * maxdist;
    const float ptx = fabsf( ::fmaf( csbp, maxdist.x, -ssbp * maxdist.y ) );
    const float pty = fabsf( ::fmaf( csbp, maxdist.y,  ssbp * maxdist.x ) );

    const float bsz = 2.0f * ( fabsf(csbp) + fabsf(ssbp) );

    const int   xmin = max(1,          (int)floorf(x - ptx - bsz));
    const int   ymin = max(1,          (int)floorf(y - pty - bsz));
    const int   xmax = min(width - 2,  (int)floorf(x + ptx + bsz));
    const int   ymax = min(height - 2, (int)floorf(y + pty + bsz));
    __shared__ float dpt[4][128];
    for( int i=threadIdx.x; i<128; i+=blockDim.x )
    {
        dpt[0][i] = 0.0f;
        dpt[1][i] = 0.0f;
        dpt[2][i] = 0.0f;
        dpt[3][i] = 0.0f;
    }
    __syncthreads();

    // we have 32 threads in a warp, how to use them?
    const int loop = threadIdx.x >> 3;             // 0 - 3
    const int xstart = ( threadIdx.x & 0x7 );
    const int xstep  = 8;

    for( int pix_y = ymin+loop; popsift::any(pix_y <= ymax); pix_y += 4 )
    {
        for( int pix_x = xmin+xstart; popsift::any(pix_x <= xmax); pix_x+=xstep )
        {
            if( ( pix_y <= ymax ) && ( pix_x <= xmax ) )
            {
                // d : distance from keypoint
                const float2 d = make_float2( pix_x - x, pix_y - y );

                // n : normalized distance from keypoint
                const float2 n = make_float2( ::fmaf( crsbp, d.x,  srsbp * d.y ),
                                              ::fmaf( crsbp, d.y, -srsbp * d.x ) ); 

                float mod;
                float th;

                get_gradiant( mod, th, pix_x, pix_y, layer_tex, level );

                mod /= 2; // Our mod is double that of vlfeat. Huh.

                const float  ww = __expf( -scalbnf(n.x*n.x + n.y*n.y, -3));

                th -= ang;
                while( th > M_PI2 ) th -= M_PI2;
                while( th < 0.0f  ) th += M_PI2;

                const float nt = 8.0f * th / M_PI2;

                // neighbouring tile on the lower side: -2, -1, 0 or 1
                // (must use floorf because casting rounds towards zero
                const int3 t0 = make_int3( (int)floorf(n.x - 0.5f),
                                        (int)floorf(n.y - 0.5f),
                                        (int)nt );
                const float wgt_x = - ( n.x - ( t0.x + 0.5f ) );
                const float wgt_y = - ( n.y - ( t0.y + 0.5f ) );
                const float wgt_t = - ( nt  - t0.z );

                for( int tx=0; tx<2; tx++ )
                {
                    for( int ty=0; ty<2; ty++ )
                    {
                        for( int tt=0; tt<2; tt++ )
                        {
                            if( ( t0.y + ty >= -2 ) &&
                                ( t0.y + ty <   2 ) &&
                                ( t0.x + tx >= -2 ) &&
                                ( t0.x + tx <   2 ) )
                            {
                                float i_wgt_x = ( tx == 0 ) ? 1.0f + wgt_x : wgt_x;
                                float i_wgt_y = ( ty == 0 ) ? 1.0f + wgt_y : wgt_y;
                                float i_wgt_t = ( tt == 0 ) ? 1.0f + wgt_t : wgt_t;

                                i_wgt_x = fabsf( i_wgt_x );
                                i_wgt_y = fabsf( i_wgt_y );
                                i_wgt_t = fabsf( i_wgt_t );

                                const float val = ww
                                                * mod
                                                * i_wgt_x
                                                * i_wgt_y
                                                * i_wgt_t;

                                const int offset =  80
                                                + ( t0.y + ty ) * 32
                                                + ( t0.x + tx ) * 8
                                                + ( t0.z + tt ) % 8;

                                atomicAdd( &dpt[loop][offset], val );
                            }
                        }
                    }
                }
            }
            __syncthreads();
        }
    }

    for( int i=threadIdx.x; i<128; i+=blockDim.x )
    {
        float f = dpt[0][i] + dpt[1][i] + dpt[2][i] + dpt[3][i];
        features[i] = f;
    }
}

__global__ void ext_desc_vlfeat(int octave, hipTextureObject_t layer_tex, int w, int h)
{           
    const int   o_offset =  dct.ori_ps[octave] + blockIdx.x;
    Descriptor* desc     = &dbuf.desc           [o_offset];
    const int   ext_idx  =  dobuf.feat_to_ext_map[o_offset];
    Extremum*   ext      =  dobuf.extrema + ext_idx;

    const int   ext_base =  ext->idx_ori;
    const int   ori_num  =  o_offset - ext_base;
    const float ang      =  ext->orientation[ori_num];

    ext_desc_vlfeat_sub( ang,
                         ext,
                         desc->features,
                         layer_tex,
                         w,
                         h );
}

namespace popsift
{

bool start_ext_desc_vlfeat( const int octave, Octave& oct_obj )
{
    dim3 block;
    dim3 grid;
    grid.x = hct.ori_ct[octave];
    grid.y = 1;
    grid.z = 1;

    if( grid.x == 0 ) return false;

    block.x = 32;
    block.y = 1;
    block.z = 1;

    size_t shared_size = 4 * 128 * sizeof(float);

    ext_desc_vlfeat
        <<<grid,block,shared_size,oct_obj.getStream()>>>
        ( octave,
          oct_obj.getDataTexPoint( ),
          oct_obj.getWidth(),
          oct_obj.getHeight() );

    POP_SYNC_CHK;

    return true;
}

}; // namespace popsift

