#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "sift_pyramid.h"
#include "sift_constants.h"
#include "gauss_filter.h"
#include "common/debug_macros.h"
#include "common/assist.h"
#include "common/clamp.h"
#include "s_pyramid_build_aa.h"
#include "s_pyramid_build_ai.h"
#include "s_pyramid_build_ra.h"

#include <iostream>
#include <stdio.h>

/* It makes no sense whatsoever to change this value */
#define PREV_LEVEL 3

using std::cout;
using std::cerr;
using std::endl;

namespace popsift {

namespace gauss {

__global__
void get_by_2_interpolate( hipTextureObject_t src_data,
                           const int           src_level,
                           hipSurfaceObject_t dst_data,
                           const int           dst_w,
                           const int           dst_h )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const float val = readTex( src_data, 2.0f * idx + 1.0f, 2.0f * idy + 1.0f, src_level );

    surf2DLayeredwrite( val, dst_data, idx*4, idy, 0, hipBoundaryModeZero ); // dst_data.ptr(idy)[idx] = val;
}

__global__
void get_by_2_pick_every_second( hipTextureObject_t src_data,
                                 const int           src_w,
                                 const int           src_h,
                                 const int           src_level,
                                 hipSurfaceObject_t dst_data,
                                 const int           dst_w,
                                 const int           dst_h )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const int read_x = clamp( idx << 1, 0, src_w );
    const int read_y = clamp( idy << 1, 0, src_h );

    const float val = readTex( src_data, read_x, read_y, src_level );

    surf2DLayeredwrite( val, dst_data, idx*4, idy, 0, hipBoundaryModeZero ); // dst_data.ptr(idy)[idx] = val;
}


__global__
void make_dog( hipTextureObject_t src_data,
               hipSurfaceObject_t dog_data,
               const int           w,
               const int           h,
               const int           max_level )
{
    const int idx   = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy   = blockIdx.y * blockDim.y + threadIdx.y;

    float a = readTex( src_data, idx, idy, 0 );
    for( int level=0; level<max_level-1; level++ )
    {
        const float b = readTex( src_data, idx, idy, level+1 );

        surf2DLayeredwrite( b-a, dog_data, idx*4, idy, level, hipBoundaryModeZero );
        a = b;
    }
}

} // namespace gauss

__host__
inline void Pyramid::horiz_from_input_image( const Config& conf, ImageBase* base, int octave, hipStream_t stream )
{
    Octave&   oct_obj = _octaves[octave];

    const int width   = oct_obj.getWidth();
    const int height  = oct_obj.getHeight();

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    const Config::SiftMode& mode = conf.getSiftMode();
    float shift  = 0.5f;

    if( octave == 0 && ( mode == Config::PopSift || mode == Config::VLFeat ) ) {
        shift  = 0.5f * powf( 2.0f, conf.getUpscaleFactor() - octave );
    }

    gauss::normalizedSource::horiz
        <<<grid,block,0,stream>>>
        ( base->getInputTexture(),
          oct_obj.getIntermediateSurface(),
          width,
          height,
          octave,
          shift );

    POP_SYNC_CHK;
}

__host__
inline void Pyramid::horiz_level_from_input_image( const Config& conf, ImageBase* base, int octave, int level, hipStream_t stream )
{
    if( octave != 0 )
    {
        POP_FATAL( "Unsupported parameter octave != 0" );
    }

    Octave&   oct_obj = _octaves[octave];

    const int width   = oct_obj.getWidth();
    const int height  = oct_obj.getHeight();

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    const Config::SiftMode& mode = conf.getSiftMode();
    float shift  = 0.5f;

    if( octave == 0 && ( mode == Config::PopSift || mode == Config::VLFeat ) ) {
        shift  = 0.5f * powf( 2.0f, conf.getUpscaleFactor() - octave );
    }

    gauss::normalizedSource::horiz_level
        <<<grid,block,0,stream>>>
        ( base->getInputTexture(),
          oct_obj.getIntermediateSurface(),
          width,
          height,
          octave,
          level,
          shift );

    POP_SYNC_CHK;
}

__host__
inline void Pyramid::horiz_all_from_input_image( const Config& conf, ImageBase* base, int octave, int startlevel, int maxlevel, hipStream_t stream )
{
    if( octave != 0 )
    {
        POP_FATAL( "Unsupported parameter octave != 0" );
    }

    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    const Config::SiftMode& mode = conf.getSiftMode();
    float shift  = 0.5f;

    if( mode == Config::PopSift || mode == Config::VLFeat ) {
        shift  = 0.5f * powf( 2.0f, conf.getUpscaleFactor() );
    }

    gauss::normalizedSource::horiz_all
        <<<grid,block,0,stream>>>
        ( base->getInputTexture(),
          oct_obj.getIntermediateSurface( ),
          width,
          height,
          shift,
          maxlevel );

    POP_SYNC_CHK;
}


__host__
inline void Pyramid::downscale_from_prev_octave( int octave, hipStream_t stream, Config::SiftMode mode )
{
    Octave&      oct_obj = _octaves[octave];
    Octave& prev_oct_obj = _octaves[octave-1];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 h_block( 64, 2 );
    dim3 h_grid;
    h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
    h_grid.y = (unsigned int)grid_divide( height, h_block.y );

    switch( mode )
    {
    case Config::PopSift :
    case Config::VLFeat :
    case Config::OpenCV :
        gauss::get_by_2_pick_every_second
            <<<h_grid,h_block,0,stream>>>
            ( prev_oct_obj.getDataTexPoint( ),
              prev_oct_obj.getWidth(),
              prev_oct_obj.getHeight(),
              _levels-PREV_LEVEL,
              oct_obj.getDataSurface( ),
              oct_obj.getWidth(),
              oct_obj.getHeight() );

        POP_SYNC_CHK;
        break;
    default :
        gauss::get_by_2_interpolate
            <<<h_grid,h_block,0,stream>>>
            ( prev_oct_obj.getDataTexLinear( ).tex,
              _levels-PREV_LEVEL,
              oct_obj.getDataSurface( ),
              oct_obj.getWidth(),
              oct_obj.getHeight() );

        POP_SYNC_CHK;
        break;
    }
}

__host__
inline void Pyramid::horiz_from_prev_level( int octave, int level, hipStream_t stream, GaussTableChoice useInterpolatedGauss )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    switch( useInterpolatedGauss )
    {
    case Interpolated_FromPrevious :
        {
            dim3 block( 128, 1 );
            dim3 grid;
            grid.x  = grid_divide( width,  128 );
            grid.y  = height;

            gauss::absoluteSourceInterpolated::horiz
                <<<grid,block,0,stream>>>
                ( oct_obj.getDataTexLinear( ).tex,
                  oct_obj.getIntermediateSurface( ),
                  level );
        }
        break;
    case NotInterpolated_FromPrevious :
        {
            dim3 block( 32,  8 ); // most stable good perf on GTX 980 TI
            // similar speed: dim3 block( 32,  4 ); dim3 block( 32,  3 ); dim3 block( 32,  2 );

            dim3 grid;
            grid.x  = grid_divide( width,  32 );
            grid.y  = grid_divide( height, block.y );

            gauss::absoluteSource::horiz
                <<<grid,block,0,stream>>>
                ( oct_obj.getDataTexPoint( ),
                  oct_obj.getIntermediateSurface( ),
                  level );
        }
        break;
    case Interpolated_FromFirst :
    case NotInterpolated_FromFirst :
        POP_FATAL( "Case horizontal Gauss filtering from first level makes not sense in case horizontal Gauss filter from previous level" );
        break;
    default :
        POP_FATAL( "Missing case in horizontal Gauss filter from previous level" );
        break;
    }
    POP_SYNC_CHK;
}

__host__
inline void Pyramid::vert_from_interm( int octave, int level, hipStream_t stream, GaussTableChoice useInterpolatedGauss )
{
    Octave& oct_obj = _octaves[octave];

    /* waiting for any events is not necessary, it's in the same stream as horiz
     */

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    switch( useInterpolatedGauss )
    {
    case Interpolated_FromPrevious :
        {
            dim3 block( 4, 32 );
            dim3 grid;
            grid.x = (unsigned int)grid_divide( width,  block.y );
            grid.y = (unsigned int)grid_divide( height, block.x );

            gauss::absoluteSourceInterpolated::vert
                <<<grid,block,0,stream>>>
                ( oct_obj.getIntermDataTexLinear( ).tex,
                  oct_obj.getDataSurface( ),
                  level );
        }
        break;
    case Interpolated_FromFirst :
        {
            dim3 block( 4, 32 );
            dim3 grid;
            grid.x = (unsigned int)grid_divide( width,  block.y );
            grid.y = (unsigned int)grid_divide( height, block.x );

            gauss::absoluteSourceInterpolated::vert_abs0
                <<<grid,block,0,stream>>>
                ( oct_obj.getIntermDataTexLinear( ).tex,
                  oct_obj.getDataSurface( ),
                  level );
        }
        break;
    case NotInterpolated_FromPrevious :
        {
            dim3 block( 64, 2 );
            dim3 grid;
            grid.x = (unsigned int)grid_divide( width,  block.x );
            grid.y = (unsigned int)grid_divide( height, block.y );

            gauss::absoluteSource::vert
                <<<grid,block,0,stream>>>
                ( oct_obj.getIntermDataTexPoint( ),
                  oct_obj.getDataSurface( ),
                  level );
        }
        break;
    case NotInterpolated_FromFirst :
        {
            dim3 block( 64, 2 );
            dim3 grid;
            grid.x = (unsigned int)grid_divide( width,  block.x );
            grid.y = (unsigned int)grid_divide( height, block.y );

            gauss::absoluteSource::vert_abs0
                <<<grid,block,0,stream>>>
                ( oct_obj.getIntermDataTexPoint( ),
                  oct_obj.getDataSurface( ),
                  level );
        }
        break;
    default :
        {
            POP_FATAL( "Missing case in vertical Gauss filter from intermediate buffer" );
        }
        break;
    }
    POP_SYNC_CHK;
}

__host__
inline void Pyramid::vert_all_from_interm( int octave, int start_level, int max_level, hipStream_t stream, GaussTableChoice useInterpolatedGauss )
{
    Octave& oct_obj = _octaves[octave];

    /* waiting for any events is not necessary, it's in the same stream as horiz
     */

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    switch( useInterpolatedGauss )
    {
    case Interpolated_FromFirst :
        {
            dim3 block( 4, 32 );
            dim3 grid;
            grid.x = (unsigned int)grid_divide( width,  block.y );
            grid.y = (unsigned int)grid_divide( height, block.x );

            gauss::absoluteSourceInterpolated::vert_all_abs0
                <<<grid,block,0,stream>>>
                ( oct_obj.getIntermDataTexLinear( ).tex,
                  oct_obj.getDataSurface( ),
                  start_level,
                  max_level );
        }
        break;
    case NotInterpolated_FromFirst :
        {
            dim3 block( 64, 2 );
            dim3 grid;
            grid.x = (unsigned int)grid_divide( width,  block.x );
            grid.y = (unsigned int)grid_divide( height, block.y );

            gauss::absoluteSource::vert_all_abs0
                <<<grid,block,0,stream>>>
                ( oct_obj.getIntermDataTexPoint( ),
                  oct_obj.getDataSurface( ),
                  start_level,
                  max_level );
        }
        break;
    case Interpolated_FromPrevious :
    case NotInterpolated_FromPrevious :
        POP_FATAL( "Case horizontal Gauss filtering from intermediate level makes not sense in case vertial-all Gauss filter from previous level" );
        break;
    default :
        POP_FATAL( "Missing case in vertical-all Gauss filter from intermediate buffer" );
        break;
    }
    POP_SYNC_CHK;
}

__host__
inline void Pyramid::dogs_from_blurred( int octave, int max_level, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 1024, 1 );
    dim3 grid;
    grid.x = grid_divide( width,  block.x );
    grid.y = grid_divide( height, block.y );
    grid.z = 1;

    gauss::make_dog
        <<<grid,block,0,stream>>>
        ( oct_obj.getDataTexPoint( ),
          oct_obj.getDogSurface( ),
          oct_obj.getWidth(),
          oct_obj.getHeight(),
          max_level );
    POP_SYNC_CHK;
}

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_pyramid( const Config& conf, ImageBase* base )
{
#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

    hipDeviceSynchronize();

    for( uint32_t octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];
        hipStream_t stream  = oct_obj.getStream();

        if( ( conf.getScalingMode() == Config::ScaleDirect ) &&
            ( conf.getGaussMode() == Config::Fixed9 || conf.getGaussMode() == Config::Fixed15 ) ) {
            if( octave == 0 ) {
                make_octave( conf, base, oct_obj, stream, true );
            } else {
                horiz_from_input_image( conf, base, octave, stream );
                vert_from_interm( octave, 0, stream, NotInterpolated_FromPrevious );
                make_octave( conf, base, oct_obj, stream, false );
            }
        } else if( conf.getGaussMode() == Config::Fixed9 || conf.getGaussMode() == Config::Fixed15 ) {
            if( octave == 0 ) {
                make_octave( conf, base, oct_obj, stream, true );
            } else {
                Octave& prev_oct_obj = _octaves[octave-1];
                cuda::event_wait( prev_oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );

                downscale_from_prev_octave( octave, stream, conf.getSiftMode() );
                make_octave( conf, base, oct_obj, stream, false );
            }

            cuda::event_record( oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );
        } else if( conf.getScalingMode() == Config::ScaleDirect ) {
            GaussTableChoice useGauss = ( conf.getGaussMode() == Config::VLFeat_Relative ) ? Interpolated_FromPrevious
                                                                                           : NotInterpolated_FromPrevious;
            for( int level=0; level<_levels; level++ ) {
                const int width  = oct_obj.getWidth();
                const int height = oct_obj.getHeight();

                if( level == 0 )
                {
                    horiz_from_input_image( conf, base, octave, stream );
                    vert_from_interm( octave, level, stream, useGauss );
                }
                else
                {
                    horiz_from_prev_level( octave, level, stream, useGauss );
                    vert_from_interm( octave, level, stream, useGauss );
                }
            }
        } else if( conf.getGaussMode() == Config::VLFeat_Relative ) {
            for( int level=0; level<_levels; level++ )
            {
                if( level == 0 )
                {
                    if( octave == 0 )
                    {
                        horiz_from_input_image( conf, base, 0, stream );
                        vert_from_interm( octave, 0, stream, Interpolated_FromPrevious );
                    }
                    else
                    {
                        Octave& prev_oct_obj = _octaves[octave-1];
                        cuda::event_wait( prev_oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );

                        downscale_from_prev_octave( octave, stream, conf.getSiftMode() );
                    }
                }
                else
                {
                    horiz_from_prev_level( octave, level, stream, Interpolated_FromPrevious );
                    vert_from_interm( octave, level, stream, Interpolated_FromPrevious );

                    if( level == _levels - PREV_LEVEL ) {
                        cuda::event_record( oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );
                    }
                }
            }
        } else if( octave == 0 && conf.getGaussMode() == Config::VLFeat_Relative_All ) {
            horiz_all_from_input_image( conf, base, octave, 0, _levels, stream );
            vert_all_from_interm( octave, 0, _levels, stream, NotInterpolated_FromFirst );
            cuda::event_record( oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );
        } else {
            for( int level=0; level<_levels; level++ )
            {
                if( level == 0 )
                {
                    if( octave == 0 )
                    {
                        horiz_from_input_image( conf, base, 0, stream );
                        vert_from_interm( octave, 0, stream, NotInterpolated_FromPrevious );
                    }
                    else
                    {
                        Octave& prev_oct_obj = _octaves[octave-1];
                        cuda::event_wait( prev_oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );

                        downscale_from_prev_octave( octave, stream, conf.getSiftMode() );
                    }
                }
                else
                {
                    horiz_from_prev_level( octave, level, stream, NotInterpolated_FromPrevious );
                    vert_from_interm( octave, level, stream, NotInterpolated_FromPrevious );

                    if( level == _levels - PREV_LEVEL ) {
                        cuda::event_record( oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );
                    }
                }
            }
        }
    }
    // for( int octave=_num_octaves-1; octave>=0; octave-- )
    for( int octave=0; octave<_num_octaves; octave++ )
    {
        if( conf.getGaussMode() == Config::Fixed9 || conf.getGaussMode() == Config::Fixed15 ) {
        } else {
            Octave&      oct_obj = _octaves[octave];
            hipStream_t stream  = oct_obj.getStream();
            dogs_from_blurred( octave, _levels, stream );
        }
    }
    for( int octave=0; octave<_num_octaves; octave++ )
    // for( int octave=_num_octaves-1; octave>=0; octave-- )
    {
        Octave&      oct_obj = _octaves[octave];
        hipStream_t stream  = oct_obj.getStream();
        hipStreamSynchronize( stream );
    }
}

} // namespace popsift

