#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "common/assist.h"
#include "common/debug_macros.h"
#include "features.h"
#include "sift_extremum.h"

#include <hip/hip_math_constants.h>

#include <cerrno>
#include <cstdlib>
#include <iomanip>
#include <iostream>

using namespace std;

namespace popsift {

/*************************************************************
 * FeaturesBase
 *************************************************************/

FeaturesBase::FeaturesBase( )
    : _num_ext( 0 )
    , _num_ori( 0 )
{ }

FeaturesBase::~FeaturesBase( ) = default;

/*************************************************************
 * FeaturesHost
 *************************************************************/

FeaturesHost::FeaturesHost( )
    : _ext( nullptr )
    , _ori( nullptr )
{ }

FeaturesHost::FeaturesHost( int num_ext, int num_ori )
    : _ext( nullptr )
    , _ori( nullptr )
{
    reset( num_ext, num_ori );
}

FeaturesHost::~FeaturesHost( )
{
    memalign_free( _ext );
    memalign_free( _ori );
}

void FeaturesHost::reset( int num_ext, int num_ori )
{
    if( _ext != nullptr ) { free( _ext ); _ext = nullptr; }
    if( _ori != nullptr ) { free( _ori ); _ori = nullptr; }

    _ext = (Feature*)memalign( getPageSize(), num_ext * sizeof(Feature) );
    if( _ext == nullptr ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime error:" << endl
             << "    Failed to (re)allocate memory for downloading " << num_ext << " features" << endl;
        if( errno == EINVAL ) cerr << "    Alignment is not a power of two." << endl;
        if( errno == ENOMEM ) cerr << "    Not enough memory." << endl;
        exit( -1 );
    }
    _ori = (Descriptor*)memalign( getPageSize(), num_ori * sizeof(Descriptor) );
    if( _ori == nullptr ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime error:" << endl
             << "    Failed to (re)allocate memory for downloading " << num_ori << " descriptors" << endl;
        if( errno == EINVAL ) cerr << "    Alignment is not a power of two." << endl;
        if( errno == ENOMEM ) cerr << "    Not enough memory." << endl;
        exit( -1 );
    }

    setFeatureCount( num_ext );
    setDescriptorCount( num_ori );
}

void FeaturesHost::pin( )
{
    hipError_t err;
    err = hipHostRegister( _ext, getFeatureCount() * sizeof(Feature), 0 );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime warning:" << endl
             << "    Failed to register feature memory in CUDA." << endl
             << "    Features count: " << getFeatureCount() << endl
             << "    Memory size requested: " << getFeatureCount() * sizeof(Feature) << endl
             << "    " << hipGetErrorString(err) << endl;
    }
    err = hipHostRegister( _ori, getDescriptorCount() * sizeof(Descriptor), 0 );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime warning:" << endl
             << "    Failed to register descriptor memory in CUDA." << endl
             << "    Descriptors count: " << getDescriptorCount() << endl
             << "    Memory size requested: " << getDescriptorCount() * sizeof(Descriptor) << endl
             << "    " << hipGetErrorString(err) << endl;
    }
}

void FeaturesHost::unpin( )
{
    hipHostUnregister( _ext );
    hipHostUnregister( _ori );
}

void FeaturesHost::print( std::ostream& ostr, bool write_as_uchar, bool with_orientation ) const
{
    for( int i=0; i<size(); i++ ) {
        _ext[i].print( ostr, write_as_uchar, with_orientation );
    }
}

std::ostream& operator<<( std::ostream& ostr, const FeaturesHost& feature )
{
    feature.print( ostr, false, false );
    return ostr;
}

/*************************************************************
 * FeaturesDev
 *************************************************************/

FeaturesDev::FeaturesDev( )
    : _ext( nullptr )
    , _ori( nullptr )
    , _rev( nullptr )
{ }

FeaturesDev::FeaturesDev( int num_ext, int num_ori )
    : _ext( nullptr )
    , _ori( nullptr )
    , _rev( nullptr )
{
    reset( num_ext, num_ori );
}

FeaturesDev::~FeaturesDev( )
{
    hipFree( _ext );
    hipFree( _ori );
    hipFree( _rev );
}

void FeaturesDev::reset( int num_ext, int num_ori )
{
    if( _ext != nullptr ) { hipFree( _ext ); _ext = nullptr; }
    if( _ori != nullptr ) { hipFree( _ori ); _ori = nullptr; }
    if( _rev != nullptr ) { hipFree( _rev ); _rev = nullptr; }

    _ext = popsift::cuda::malloc_mgdT<Feature>   ( num_ext, __FILE__, __LINE__ );
    _ori = popsift::cuda::malloc_mgdT<Descriptor>( num_ori, __FILE__, __LINE__ );
    _rev = popsift::cuda::malloc_mgdT<int>       ( num_ori, __FILE__, __LINE__ );

    setFeatureCount( num_ext );
    setDescriptorCount( num_ori );
}

__device__ inline float
l2_in_t0( const float4* lptr, const float4* rptr )
{
    const float4  lval = lptr[threadIdx.x];
    const float4  rval = rptr[threadIdx.x];
    const float4  mval = make_float4( lval.x - rval.x,
			              lval.y - rval.y,
			              lval.z - rval.z,
			              lval.w - rval.w );
    float   res = mval.x * mval.x
	        + mval.y * mval.y
	        + mval.z * mval.z
	        + mval.w * mval.w;
    res += shuffle_down( res, 16 );
    res += shuffle_down( res,  8 );
    res += shuffle_down( res,  4 );
    res += shuffle_down( res,  2 );
    res += shuffle_down( res,  1 );
    return res;
}

__global__ void
compute_distance( int3* match_matrix, Descriptor* l, int l_len, Descriptor* r, int r_len )
{
    if( blockIdx.x >= l_len ) return;
    const int idx = blockIdx.x;

    float match_1st_val = HIP_INF_F;
    float match_2nd_val = HIP_INF_F;
    int   match_1st_idx = 0;
    int   match_2nd_idx = 0;

    const float4* lptr = (const float4*)( &l[idx] );

    for( int i=0; i<r_len; i++ )
    {
        const float4* rptr = (const float4*)( &r[i] );

        const float   res  = l2_in_t0( lptr, rptr );

        if( threadIdx.x == 0 )
        {
            if( res < match_1st_val )
            {
                match_2nd_val = match_1st_val;
                match_2nd_idx = match_1st_idx;
                match_1st_val = res;
                match_1st_idx = i;
            }
            else if( res < match_2nd_val )
            {
                match_2nd_val = res;
                match_2nd_idx = i;
            }
        }
        __syncthreads();
    }

    if( threadIdx.x == 0 )
    {
        bool accept = ( match_1st_val / match_2nd_val < 0.8f );
        match_matrix[blockIdx.x] = make_int3( match_1st_idx, match_2nd_idx, accept );
    }
}

__global__ void
show_distance( int3*       match_matrix,
               Feature*    l_ext,
               Descriptor* l_ori,
               int*        l_fem,
               int         l_len,
               Feature*    r_ext,
               Descriptor* r_ori,
               int*        r_fem,
               int         r_len )
{
    for( int i=0; i<l_len; i++ )
    {
        const float4* lptr  = (const float4*)( &l_ori[i] );
        const float4* rptr1 = (const float4*)( &r_ori[match_matrix[i].x] );
        const float4* rptr2 = (const float4*)( &r_ori[match_matrix[i].y] );
	    float d1 = l2_in_t0( lptr, rptr1 );
	    float d2 = l2_in_t0( lptr, rptr2 );
	    if( threadIdx.x == 0 )
        {
            if( match_matrix[i].z )
            {
                Feature* lx = &l_ext[l_fem[i]];
                Feature* rx = &r_ext[r_fem[match_matrix[i].x]];
                printf( "accept feat %4d [%4d] matches feat %4d [%4d] ( 2nd feat %4d [%4d] ) dist %.3f vs %.3f"
                        " (%.1f,%.1f)-(%.1f,%.1f)\n",
                        l_fem[i], i,
                        r_fem[match_matrix[i].x], match_matrix[i].x,
                        r_fem[match_matrix[i].y], match_matrix[i].y,
                        d1, d2,
                        lx->xpos, lx->ypos, rx->xpos, rx->ypos );
            }
            else
            {
                printf( "reject feat %4d [%4d] matches feat %4d [%4d] ( 2nd feat %4d [%4d] ) dist %.3f vs %.3f\n",
                        l_fem[i], i,
                        r_fem[match_matrix[i].x], match_matrix[i].x,
                        r_fem[match_matrix[i].y], match_matrix[i].y,
                        d1, d2 );
            }
        }
        __syncthreads();
    }
}

void FeaturesDev::match( FeaturesDev* other )
{
    int l_len = getDescriptorCount( );
    int r_len = other->getDescriptorCount( );

    int3* match_matrix = popsift::cuda::malloc_devT<int3>( l_len, __FILE__, __LINE__ );

    dim3 grid;
    grid.x = l_len;
    grid.y = 1;
    grid.z = 1;
    dim3 block;
    block.x = 32;
    block.y = 1;
    block.z = 1;

    compute_distance
        <<<grid,block>>>
        ( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len );

    POP_SYNC_CHK;

    show_distance
        <<<1,32>>>
        ( match_matrix,
          getFeatures(),
          getDescriptors(),
          getReverseMap(),
          l_len,
          other->getFeatures(),
          other->getDescriptors(),
          other->getReverseMap(),
          r_len );

    POP_SYNC_CHK;

    hipFree( match_matrix );
}

int3* FeaturesDev::matchAndReturn( FeaturesDev* other )
{
    int l_len = getDescriptorCount( );
    int r_len = other->getDescriptorCount( );

    int3* match_matrix = popsift::cuda::malloc_mgdT<int3>( l_len, __FILE__, __LINE__ );

    dim3 grid;
    grid.x = l_len;
    grid.y = 1;
    grid.z = 1;
    dim3 block;
    block.x = 32;
    block.y = 1;
    block.z = 1;

    compute_distance
        <<<grid,block>>>
        ( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len );

    return match_matrix;
}

Descriptor* FeaturesDev::getDescriptor( int descIndex )
{
    return &_ori[descIndex];
}

const Descriptor* FeaturesDev::getDescriptor( int descIndex ) const
{
    return &_ori[descIndex];
}

Feature* FeaturesDev::getFeatureForDescriptor( int descIndex )
{
    return &_ext[_rev[descIndex]];
}

const Feature* FeaturesDev::getFeatureForDescriptor( int descIndex ) const
{
    return &_ext[_rev[descIndex]];
}

/*************************************************************
 * Feature
 *************************************************************/

void Feature::print( std::ostream& ostr, bool write_as_uchar, bool with_orientation ) const
{
    float sigval =  1.0f / ( sigma * sigma );

    for( int ori=0; ori<num_ori; ori++ ) {
        if( with_orientation )
        {
            float dom_ori = orientation[ori];
            // dom_ori = dom_ori / M_PI2 * 360;
            // if (dom_ori < 0) dom_ori += 360;
            if (dom_ori < 0) dom_ori += M_PI2;

            ostr << std::setprecision(6)
                 << xpos << " " << ypos << " "
                 << sigma << " "
                 << dom_ori << " ";
        }
        else
        {
            ostr << std::setprecision(6)
                 << xpos << " " << ypos << " "
                 << sigval << " 0 "
                 << sigval << " ";
        }
        if( write_as_uchar ) {
            for( int i=0; i<128; i++ ) {
                ostr << roundf(desc[ori]->features[i]) << " ";
            }
        } else {
            ostr << std::setprecision(3);
            for( int i=0; i<128; i++ ) {
                ostr << desc[ori]->features[i] << " ";
            }
            ostr << std::setprecision(6);
        }
        ostr << std::endl;
    }
}

std::ostream& operator<<( std::ostream& ostr, const Feature& feature )
{
    feature.print( ostr, false, false );
    return ostr;
}

} // namespace popsift
