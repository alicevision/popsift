#include "hip/hip_runtime.h"
/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include <fstream>

#include "sift_constants.h"
#include "popsift.h"
#include "gauss_filter.h"
#include "common/write_plane_2d.h"
#include "sift_pyramid.h"
#include "sift_extremum.h"

using namespace std;

PopSift::PopSift( const popsift::Config& config )
{
    for( int i=0; i<MAX_PIPES; i++ ) {
        _pipe[i]._inputImage = 0;
        _pipe[i]._pyramid    = 0;
    }

    configure(config);
}

PopSift::PopSift( )
{
    for( int i=0; i<MAX_PIPES; i++ ) {
        _pipe[i]._inputImage = 0;
        _pipe[i]._pyramid    = 0;
    }
}

PopSift::~PopSift()
{ }

bool PopSift::configure( const popsift::Config& config )
{
    for( int i=0; i<MAX_PIPES; i++ ) {
        if( _pipe[i]._inputImage != 0 ) {
            return false;
        }
    }

    _config = config;

    _config.levels = max( 2, config.levels );

    popsift::init_filter( _config,
                         _config.sigma,
                         _config.levels );
    popsift::init_constants(  _config.sigma,
                             _config.levels,
                             _config.getPeakThreshold(),
                             _config._edge_limit,
                             10000, // max extrema
                             _config.getNormalizationMultiplier() );
    return true;
}

bool PopSift::init( int pipe, int w, int h, bool checktime )
{
    if( _pipe[pipe]._inputImage != 0 ) return false;

    hipEvent_t start, end;
    if( checktime ) {
        hipEventCreate( &start );
        hipEventCreate( &end );
        hipDeviceSynchronize();
        hipEventRecord( start, 0 );
    }

    if( pipe < 0 && pipe >= MAX_PIPES ) {
        return false;
    }

    /* up=-1 -> scale factor=2
     * up= 0 -> scale factor=1
     * up= 1 -> scale factor=0.5
     */
    float upscaleFactor = _config.getUpscaleFactor();
    float scaleFactor = 1.0f / powf( 2.0f, -upscaleFactor );

    if( _config.octaves < 0 ) {
        int oct = _config.octaves;
        oct = max(int (floor( logf( (float)min( w, h ) )
                            / logf( 2.0f ) ) - 3.0f + scaleFactor ), 1);
        _config.octaves = oct;
    }

    _pipe[pipe]._inputImage = new popsift::Image( w, h );
    _pipe[pipe]._pyramid = new popsift::Pyramid( _config,
                                                _pipe[pipe]._inputImage,
                                                ceilf( w * scaleFactor ),
                                                ceilf( h * scaleFactor ) );

    hipDeviceSynchronize();

    if( checktime ) {
        hipEventRecord( end, 0 );
        hipEventSynchronize( end );
        float elapsedTime;
        hipEventElapsedTime( &elapsedTime, start, end );

        cerr << "Initialization of pipe " << pipe << " took " << elapsedTime << " ms" << endl;
    }

    return true;
}

void PopSift::uninit( int pipe )
{
    if( pipe < 0 && pipe >= MAX_PIPES ) return;

    delete _pipe[pipe]._inputImage;
    delete _pipe[pipe]._pyramid;

    _pipe[pipe]._inputImage = 0;
    _pipe[pipe]._pyramid    = 0;
}

popsift::Features* PopSift::execute( int                  pipe,
                                     const unsigned char* imageData,
                                     bool                 checktime )
{
    if( _pipe[pipe]._inputImage == 0 ) return 0;

    if( pipe < 0 && pipe >= MAX_PIPES ) return 0;

    hipEvent_t start, end;
    if( checktime ) {
        hipEventCreate( &start );
        hipEventCreate( &end );

        hipDeviceSynchronize();
        hipEventRecord( start, 0 );
    }

    _pipe[pipe]._inputImage->load( _config, imageData );

    popsift::Features* features = _pipe[pipe]._pyramid->find_extrema( _config, _pipe[pipe]._inputImage );

    hipDeviceSynchronize();

    if( checktime ) {
        hipEventRecord( end, 0 );
        hipEventSynchronize( end );
        float elapsedTime;
        hipEventElapsedTime( &elapsedTime, start, end );

        cerr << "Execution of pipe " << pipe << " took " << elapsedTime << " ms" << endl;
    }

    bool log_to_file = ( _config.log_mode == popsift::Config::All );
    if( log_to_file ) {
        int octaves = _pipe[pipe]._pyramid->getNumOctaves();

        for( int o=0; o<octaves; o++ ) {
            _pipe[pipe]._pyramid->download_descriptors( _config, o );
        }

        int levels  = _pipe[pipe]._pyramid->getNumLevels();

        for( int o=0; o<octaves; o++ ) {
            for( int s=0; s<levels+3; s++ ) {
                _pipe[pipe]._pyramid->download_and_save_array( "pyramid", o, s );
            }
        }
        for( int o=0; o<octaves; o++ ) {
            _pipe[pipe]._pyramid->save_descriptors( _config, "pyramid", o );
        }
    }

    return features;
}

